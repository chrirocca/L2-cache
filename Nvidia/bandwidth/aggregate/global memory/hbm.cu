
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

#define ZCOPY_THREADS  32
#define ZCOPY_DEFLEN   30000000
#define ZCOPY_ITER     10           // as in STREAM benchmark
#define NUM_SM         108           // number of SMs on device

// Macro to catch CUDA errors in CUDA runtime calls
#define CUDA_SAFE_CALL(call)                                          \
do {                                                                  \
    hipError_t err = call;                                           \
    if (hipSuccess != err) {                                         \
        fprintf (stderr, "Cuda error in file '%s' in line %i : %s.\n",\
                 __FILE__, __LINE__, hipGetErrorString(err) );       \
        exit(EXIT_FAILURE);                                           \
    }                                                                 \
} while (0)

// Macro to catch CUDA errors in kernel launches
#define CHECK_LAUNCH_ERROR()                                          \
do {                                                                  \
    /* Check synchronous errors, i.e. pre-launch */                   \
    hipError_t err = hipGetLastError();                             \
    if (hipSuccess != err) {                                         \
        fprintf (stderr, "Cuda error in file '%s' in line %i : %s.\n",\
                 __FILE__, __LINE__, hipGetErrorString(err) );       \
        exit(EXIT_FAILURE);                                           \
    }                                                                 \
    /* Check asynchronous errors, i.e. kernel failed (ULF) */         \
    err = hipDeviceSynchronize();                                    \
    if (hipSuccess != err) {                                         \
        fprintf (stderr, "Cuda error in file '%s' in line %i : %s.\n",\
                 __FILE__, __LINE__, hipGetErrorString( err) );      \
        exit(EXIT_FAILURE);                                           \
    }                                                                 \
} while (0)

#if defined(_WIN32)
#if !defined(WIN32_LEAN_AND_MEAN)
#define WIN32_LEAN_AND_MEAN
#endif
#include <windows.h>
double second (void)
{
    LARGE_INTEGER t;
    static double oofreq;
    static int checkedForHighResTimer;
    static BOOL hasHighResTimer;

    if (!checkedForHighResTimer) {
        hasHighResTimer = QueryPerformanceFrequency (&t);
        oofreq = 1.0 / (double)t.QuadPart;
        checkedForHighResTimer = 1;
    }
    if (hasHighResTimer) {
        QueryPerformanceCounter (&t);
        return (double)t.QuadPart * oofreq;
    } else {
        return (double)GetTickCount() * 1.0e-3;
    }
}
#elif defined(__linux__) || defined(__APPLE__)
#include <stddef.h>
#include <sys/time.h>
double second (void)
{
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return (double)tv.tv_sec + (double)tv.tv_usec * 1.0e-6;
}
#else
#error unsupported platform
#endif

__global__ void zcopy (const double2 * __restrict__ src, 
                       double2 * __restrict__ dst, int len)
{
    int stride = gridDim.x * blockDim.x;
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    for (int i = tid; i < len; i += stride) {
        dst[i] = src[i];
    }
}    

struct zcopyOpts {
    int len;
};

static int processArgs (int argc, char *argv[], struct zcopyOpts *opts)
{
    int error = 0;
    memset (opts, 0, sizeof(*opts));
    while (argc) {
        if (*argv[0] == '-') {
            switch (*(argv[0]+1)) {
            case 'n':
                opts->len = atol(argv[0]+2);
                break;
            default:
                fprintf (stderr, "Unknown switch '%c%s'\n", '-', argv[0]+1);
                error++;
                break;
            }
        }
        argc--;
        argv++;
    }
    return error;
}

int main (int argc, char *argv[])
{
    double start, stop, elapsed, mintime;
    double2 *d_a, *d_b;
    int errors;
    struct zcopyOpts opts;

    int CTA = atoi(argv[1]);

    errors = processArgs (argc, argv, &opts);
    if (errors) {
        return EXIT_FAILURE;
    }
    opts.len = (opts.len) ? opts.len : ZCOPY_DEFLEN;

    {
        size_t stacksize;
        CUDA_SAFE_CALL (hipDeviceGetLimit (&stacksize, hipLimitStackSize));
        //printf ("stacksize = %lu\n", stacksize);
    }


    /* Allocate memory on device */
    CUDA_SAFE_CALL (hipMalloc((void**)&d_a, sizeof(d_a[0]) * opts.len));
    CUDA_SAFE_CALL (hipMalloc((void**)&d_b, sizeof(d_b[0]) * opts.len));
    
    /* Initialize device memory */
    CUDA_SAFE_CALL (hipMemset(d_a, 0x00, sizeof(d_a[0]) * opts.len)); // zero
    CUDA_SAFE_CALL (hipMemset(d_b, 0xff, sizeof(d_b[0]) * opts.len)); // NaN

    /* Compute execution configuration */
    dim3 dimBlock(ZCOPY_THREADS);
    //int threadBlocks = (opts.len + (dimBlock.x - 1)) / dimBlock.x;
    int threadBlocks = NUM_SM*CTA;
    dim3 dimGrid(threadBlocks);
    
    //printf ("zcopy: operating on vectors of %d double2s (= %.3f MB)\n", 
            //opts.len, (double)sizeof(d_a[0]) * opts.len /(1024*1024));
    //printf ("zcopy: using %d threads per block, %d blocks\n", 
            //dimBlock.x, dimGrid.x);

    mintime = fabs(log(0.0));
    for (int k = 0; k < ZCOPY_ITER; k++) {
        start = second();
        zcopy<<<dimGrid,dimBlock>>>(d_a, d_b, opts.len);
        CHECK_LAUNCH_ERROR();
        stop = second();
        elapsed = stop - start;
        if (elapsed < mintime) mintime = elapsed;
    }
    //printf ("zcopy: mintime = %.3f msec  throughput = %.2f GB/sec\n",
            //1.0e3 * mintime, (2.0e-9 * sizeof(d_a[0]) * opts.len) / mintime);
    printf("%.2f\n", (2.0e-9 * sizeof(d_a[0]) * opts.len) / mintime);

    CUDA_SAFE_CALL (hipFree(d_a));
    CUDA_SAFE_CALL (hipFree(d_b));

    return EXIT_SUCCESS;
}