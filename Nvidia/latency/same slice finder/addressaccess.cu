
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define NUM_SM 80
#define BLOCK_SIZE 32 // related with tid*N°
#define S_SIZE ((6*1024)*1024)/4 // must be smaller than L2 cache size
#define ITERATION 2

typedef unsigned int uint;

#define cudaCheckError() {                                          \
    hipError_t e=hipGetLastError();                                 \
    if(e!=hipSuccess) {                                              \
        printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));           \
        exit(0); \
    }                                                                 \
}

__device__ unsigned int get_smid(void) {
    unsigned int ret;
    asm("mov.u32 %0, %smid;" : "=r"(ret) );
    return ret;
}

__global__ void kernel(unsigned int *a0, unsigned int *timing, unsigned int start_idx) {
    __shared__ unsigned int measured_latency[12288];
    unsigned int i,j,k,idx;
    unsigned int tid = threadIdx.x;
    unsigned int sm_id = get_smid();
    unsigned int start, latency;

    if(sm_id == 0) 
    {
        k = 0;
        __syncthreads();

        for (i = 0; i < ITERATION; i ++)
        {
            idx = start_idx*256;

            start = clock();

            for (j = 0; j < 2; j++){
                k += a0[idx];
                //printf("thread : %u\n", tid);
            }

            latency = clock() - start;

            a0[0] = k; // needed for forcing dependency

            if(tid == 0) {
                measured_latency[i] = latency;
                timing[i] = measured_latency[i];
            }
        }
        
    }

}
    
int main(int argc, char * argv[]) {
    unsigned int * h_arr;
    unsigned int * h_timing;
    unsigned int * d_a0;
    unsigned int * d_timing;
    int i,start_idx;
    int percentage;
    
    hipSetDevice(0);

    percentage = atoi(argv[1]);
    start_idx = atoi(argv[2]);

    h_arr = (unsigned int *)malloc(sizeof(unsigned int) * S_SIZE);
    h_timing = (unsigned int *)malloc(sizeof(unsigned int) * ITERATION);

    hipMalloc((void**)&d_a0, sizeof(unsigned int) * S_SIZE);
    cudaCheckError();

    hipMalloc((void**)&d_timing, sizeof(unsigned int) * ITERATION);
    cudaCheckError();


    for (i = 0; i < S_SIZE; i++) {
        h_arr[i] = 10;
    }

    hipMemcpy(d_a0, h_arr, sizeof(unsigned int) * S_SIZE, hipMemcpyHostToDevice);
    cudaCheckError();

    int carveout = percentage; // prefer shared memory capacity 100% of maximum
    hipFuncSetAttribute (reinterpret_cast<const void*>(kernel), hipFuncAttributePreferredSharedMemoryCarveout, carveout);
    cudaCheckError();

    kernel<<<NUM_SM, BLOCK_SIZE>>>(d_a0, d_timing,start_idx);
    cudaCheckError();
    hipDeviceSynchronize();
    cudaCheckError();

    hipMemcpy(h_timing, d_timing, sizeof(unsigned int) * ITERATION, hipMemcpyDeviceToHost);
    cudaCheckError();

    for(int i = 0; i < ITERATION; i++)
        printf("measured latency is %u\n",h_timing[i]);

    free(h_arr);
    free(h_timing);
    hipFree(d_a0); 
    hipFree(d_timing);

    return 0;
}


